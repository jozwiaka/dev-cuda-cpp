#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void vectorAdd(float *a, float *b, float *c, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        printf("Index: %d\n", index);
        c[index] = a[index] + b[index];
    }
}

int main()
{
    int n = 1000; // Size of vectors
    size_t bytes = n * sizeof(float);

    // Allocate host memory
    float *h_a = (float *)malloc(bytes);
    float *h_b = (float *)malloc(bytes);
    float *h_c = (float *)malloc(bytes);

    // Initialize vectors
    for (int i = 0; i < n; i++)
    {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Define kernel launch configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Copy result from device to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < n; i++)
    {
        if (h_c[i] != h_a[i] + h_b[i])
        {
            std::cerr << "Error: Incorrect result at index " << i << std::endl;
            break;
        }
        std::cout << "h_c[" << i << "] = " << h_c[i] << std::endl;
    }

    std::cout << "Vector addition successful!" << std::endl;

    // Free device and host memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
